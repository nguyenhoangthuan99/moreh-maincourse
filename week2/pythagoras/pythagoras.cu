
#include <hip/hip_runtime.h>
#include <cstdio>

#define CHECK_CUDA(call)                                                 \
  do                                                                     \
  {                                                                      \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess)                                          \
    {                                                                    \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

__global__ void pythagoras(int *pa, int *pb, int *pc, int *presult)
{
  int a = *pa;
  int b = *pb;
  int c = *pc;

  if ((a * a + b * b) == c * c)
    *presult = 1;
  else
    *presult = 0;
}

int main(int argc, char *argv[])
{
  if (argc != 4)
  {
    printf("Usage: %s <num 1> <num 2> <num 3>\n", argv[0]);
    return 0;
  }

  int a = atoi(argv[1]);
  int b = atoi(argv[2]);
  int c = atoi(argv[3]);
  int result = 0;

  // TODO: 1. allocate device memory
  int *A, *B, *C, *result_cuda;
  CHECK_CUDA(hipMalloc(&A, sizeof(int)));
  CHECK_CUDA(hipMalloc(&B, sizeof(int)));
  CHECK_CUDA(hipMalloc(&C, sizeof(int)));
  CHECK_CUDA(hipMalloc(&result_cuda, sizeof(int)));

  // TODO: 2. copy data to device
  CHECK_CUDA(hipMemcpy(A, &a, sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(B, &b, sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(C, &c, sizeof(int), hipMemcpyHostToDevice));

  // TODO: 3. launch kernel
  pythagoras<<<1, 1>>>(A, B, C, result_cuda);
  CHECK_CUDA(hipGetLastError()) ;

  // TODO: 4. copy result back to host
  CHECK_CUDA(hipMemcpy(&result, result_cuda, sizeof(int), hipMemcpyDeviceToHost));
  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(result_cuda);
  if (result)
    printf("YES\n");
  else
    printf("NO\n");

  return 0;
}
