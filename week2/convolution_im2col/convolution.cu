#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include "convolution.cuh"
#include "util.h"

#include <hipblas.h>

#define BLOCKS 16
#define BLOCK_SIZE 32
static int ngpu;
static float *I_gpu, *F_gpu, *O_gpu, *BUF1_gpu, *BUF2_gpu;
#define CHECK_CUDA(call)                                                 \
  do                                                                     \
  {                                                                      \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess)                                          \
    {                                                                    \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

__global__ void im2col_kernel(float *_I, float *workspace, int N, int C, int H, int W,
                              int R, int S, int pad_h, int pad_w, int stride_h,
                              int stride_w, int dilation_h, int dilation_w)
{
  const int ON = N;
  const int OH = 1 + (H + 2 * pad_h - (((R - 1) * dilation_h) + 1)) / stride_h;
  const int OW = 1 + (W + 2 * pad_w - (((S - 1) * dilation_w) + 1)) / stride_w;
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  // int ow = tidx % OW;
  // int oh = (tidx) / OW % OH;
  // int on = tidx / OW / OH;

  const int ow = tidx % OW;
  const int oh = (tidx / OW) % OH;
  const int on = tidx / (OH * OW);
  if (ow < OW && oh < OH && on < N)
  {
#pragma unroll
    for (int c = 0; c < C; ++c)
      for (int r = 0; r < R; ++r)
        for (int s = 0; s < S; ++s)
        {
          const int h = oh * stride_h - pad_h + r * dilation_h;
          const int w = ow * stride_w - pad_w + s * dilation_w;
          if (h < 0 || h >= H || w < 0 || w >= W)
            continue;
          // float temp = ;
          // for (int on = 0; on < ON; ++on)
          // {

          workspace[((c * R * S) + (r * S) + s) * (ON * OH * OW) +
                    (on * OH * OW + oh * OW + ow)] =
              _I[on * C * H * W + c * H * W + h * W + w];
          // }
        }
  }
}

__global__ void matmul_gpu(float *_A, float *_B, float *_C, int M, int N, int K, int temp_N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int k;
  __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B1[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B2[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B3[BLOCK_SIZE][BLOCK_SIZE];
  float c0;
  float4 sum = make_float4(0, 0, 0, 0);
  // float4 sum2 = make_float4(0, 0, 0, 0);
  int boundk = (K + BLOCK_SIZE - 1) / BLOCK_SIZE;
  int aid = idy * K + threadIdx.x;
  int bid = threadIdx.y * N + idx * 4;
  for (k = 0; k < boundk; k++)
  {
    As[threadIdx.y][threadIdx.x] = (idy < M) && k * BLOCK_SIZE + threadIdx.x < K ? _A[aid] : 0;
    Bs[threadIdx.y][threadIdx.x] = (idx * 4 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid] : 0;
    B1[threadIdx.y][threadIdx.x] = (idx * 4 + 1 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid + 1] : 0;
    B2[threadIdx.y][threadIdx.x] = (idx * 4 + 2 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid + 2] : 0;
    B3[threadIdx.y][threadIdx.x] = (idx * 4 + 3 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid + 3] : 0;
    __syncthreads();

    // #pragma unroll 32
    for (int e = 0; e < BLOCK_SIZE; e++)
    {
      float a = As[threadIdx.y][e];
      sum = make_float4(sum.x + a * Bs[e][threadIdx.x], sum.y + a * B1[e][threadIdx.x], sum.z + a * B2[e][threadIdx.x], sum.w + a * B3[e][threadIdx.x]);
    }
    __syncthreads();
    aid += BLOCK_SIZE;
    bid += BLOCK_SIZE * N;
  }
  if (idy < M)
  {

    int temp_K = M, temp_OHOW = N / temp_N;
    float left = min(4, N - idx * 4);
#pragma unroll 4
    for (int i = 0; i < left; i++)
    {
      int c_index = idx * 4 + i + idy * N;
      int ohow = c_index % temp_OHOW;
      int n = c_index / temp_OHOW % temp_N;
      k = c_index / temp_OHOW / temp_N;
      int new_c_index = ((n * temp_K + k) * temp_OHOW + ohow);

      _C[new_c_index] = ((float *)(&sum))[i];
    }
  }
}
__global__ void reshape_gpu(float *_src, float *_dst, int N, int K, int OH, int OW)
{
  extern __shared__ float shared[];
  size_t chunk = OH * OW;

  const int on = blockDim.x * blockIdx.x + threadIdx.x;
  const int k = blockDim.y * blockIdx.y + threadIdx.y;

  if (on < N && k < K)
  {
    const int src_offset = on * K * chunk + k * chunk;
    const int dst_offset = on * K * chunk + k * chunk;

    for (int i = 0; i < chunk; ++i)
    {
      shared[threadIdx.x * blockDim.y + threadIdx.y + i] =
          _src[src_offset + i];
    }

    __syncthreads();

    for (int i = 0; i < chunk; ++i)
    {
      _dst[dst_offset + i] =
          shared[threadIdx.x * blockDim.y + threadIdx.y + i];
    }
  }
}
void reshape(float *_src, float *_dst, int N, int K, int OH, int OW)
{
  size_t chunk = OH * OW;
#pragma omp parallel for num_threads(32)
  for (int k = 0; k < K; ++k)
    for (int on = 0; on < N; ++on)
    {
      // for (int idx = 0; idx < K * N; idx++)
      {
        // int k = idx % K;
        // int on = idx / K;
        memcpy((void *)(_dst + ((on * K + k) * chunk)),
               (void *)(_src + ((k * N + on) * chunk)), chunk * sizeof(float));
      }
    }
}

void convolution(float *_I, float *_F, float *_O, float *_BUF1, float *_BUF2, int N,
                 int C, int H, int W, int K, int R, int S, int pad_h, int pad_w,
                 int stride_h, int stride_w, int dilation_h, int dilation_w)
{
  // Remove this line after you complete the convolution on GPU
  // naive_cpu_convolution_im2col(_I, _F, _O, _BUF1, _BUF2, N, C, H, W, K, R, S,
  //                              pad_h, pad_w, stride_h, stride_w, dilation_h,
  //                              dilation_w);
  float *I = _I, *F = _F, *O = _O, *BUF1 = _BUF1, *BUF2 = _BUF2;
  hipSetDevice(0);

  hipStream_t data_h2d_stream, data_d2h_stream, calc_im2col_stream, calc_matmul_stream;
  hipStreamCreate(&data_h2d_stream);
  hipStreamCreate(&data_d2h_stream);
  hipStreamCreate(&calc_im2col_stream);
  hipStreamCreate(&calc_matmul_stream);
  hipEvent_t events_data[BLOCKS], events_im2col_cals[BLOCKS], events_matmul_cals[BLOCKS];

  for (int i = 0; i < BLOCKS; i++)
  {
    hipEventCreate(&events_data[i]);
    hipEventCreate(&events_im2col_cals[i]);
    hipEventCreate(&events_matmul_cals[i]);
  }

  int Nbegin[BLOCKS], Nend[BLOCKS];
  for (size_t i = 0; i < BLOCKS; i++)
  {
    Nbegin[i] = N / BLOCKS * i;
    Nend[i] = N / BLOCKS * (i + 1);
    if (i == BLOCKS - 1)
      Nend[i] = N;
  }

  const int OH = 1 + (H + 2 * pad_h - (((R - 1) * dilation_h) + 1)) / stride_h;
  const int OW = 1 + (W + 2 * pad_w - (((S - 1) * dilation_w) + 1)) / stride_w;

  hipMemcpyAsync(F_gpu, _F, sizeof(float) * K * C * R * S, hipMemcpyHostToDevice, data_h2d_stream);

  for (int i = 0; i < BLOCKS; i++)
  {
    hipMemcpyAsync(&I_gpu[Nbegin[i] * C * H * W], &_I[Nbegin[i] * C * H * W], sizeof(float) * (Nend[i] - Nbegin[i]) * C * H * W, hipMemcpyHostToDevice, data_h2d_stream);
    hipEventRecord(events_data[i], data_h2d_stream);
  }

  for (int i = 0; i < BLOCKS; i++)
  {
    dim3 blockDimIm2Col(768);
    dim3 gridDimIm2Col((OH * OW * (Nend[i] - Nbegin[i]) + blockDimIm2Col.x - 1) / blockDimIm2Col.x);
    hipStreamWaitEvent(calc_im2col_stream, events_data[i]);
    im2col_kernel<<<gridDimIm2Col, blockDimIm2Col, 0, calc_im2col_stream>>>(&I_gpu[Nbegin[i] * C * H * W], &BUF1_gpu[Nbegin[i] * C * R * S * OH * OW],
                                                                            (Nend[i] - Nbegin[i]), C, H, W,
                                                                            R, S, pad_h, pad_w, stride_h,
                                                                            stride_w, dilation_h, dilation_w);
    hipEventRecord(events_im2col_cals[i], calc_im2col_stream);
    // CHECK_CUDA(hipDeviceSynchronize());
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 gridDim((((Nend[i] - Nbegin[i]) * OH * OW + 3) / 4 + blockDim.x - 1) / blockDim.x, (K + blockDim.y - 1) / blockDim.y, 1);
    hipStreamWaitEvent(calc_matmul_stream, events_im2col_cals[i]);
    matmul_gpu<<<gridDim, blockDim, 0, calc_matmul_stream>>>(F_gpu, &BUF1_gpu[Nbegin[i] * C * R * S * OH * OW],
                                                             &BUF2_gpu[Nbegin[i] * K * OH * OW],
                                                             K, (Nend[i] - Nbegin[i]) * OH * OW, C * R * S,
                                                             (Nend[i] - Nbegin[i]));
    hipEventRecord(events_matmul_cals[i], calc_matmul_stream);

    hipStreamWaitEvent(data_d2h_stream, events_matmul_cals[i]);
    hipMemcpyAsync(&_O[Nbegin[i] * K * OH * OW], &BUF2_gpu[Nbegin[i] * K * OH * OW], sizeof(float) * K * (Nend[i] - Nbegin[i]) * OH * OW, hipMemcpyDeviceToHost, data_d2h_stream);
  }

  CHECK_CUDA(hipDeviceSynchronize());
  // double start = get_time();
  // reshape(BUF2, O, N, K, OH, OW);
  // double end = get_time();
  // printf("\nreshape time: %lf\n", end - start);
  // CHECK_CUDA(hipDeviceSynchronize());
}

void convolution_init(int N, int C, int H, int W, int K, int R, int S,
                      int pad_h, int pad_w, int stride_h, int stride_w,
                      int dilation_h, int dilation_w)
{

  const int ON = N;
  const int OC = K;
  const int OH = 1 + (H + 2 * pad_h - (((R - 1) * dilation_h) + 1)) / stride_h;
  const int OW = 1 + (W + 2 * pad_w - (((S - 1) * dilation_w) + 1)) / stride_w;
  CHECK_CUDA(hipMalloc(&I_gpu, sizeof(float) * N * C * H * W));
  // CHECK_CUDA(hipMalloc(&O_gpu, sizeof(float) * ON * OH * OW * OC));
  CHECK_CUDA(hipMalloc(&F_gpu, sizeof(float) * K * C * R * S));
  CHECK_CUDA(hipMalloc(&BUF1_gpu, sizeof(float) * C * R * S * ON * OH * OW));
  CHECK_CUDA(hipMalloc(&BUF2_gpu, sizeof(float) * K * N * OH * OW));
  CHECK_CUDA(hipDeviceSynchronize());
}

void convolution_cleanup(float *_I, float *_F, float *_O, int N, int C, int H,
                         int W, int K, int R, int S, int pad_h, int pad_w,
                         int stride_h, int stride_w, int dilation_h,
                         int dilation_w)
{
  hipFree(I_gpu);
  hipFree(F_gpu);
  // hipFree(O_gpu);
  hipFree(BUF1_gpu);
  hipFree(BUF2_gpu);
  CHECK_CUDA(hipDeviceSynchronize());
}
// hipblasHandle_t handle;
// hipblasCreate(&handle);
// const float alpha = 1.0f, beta = 0.0f;
// hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N * OH * OW, K, C * R * S,
//              &alpha, BUF1_gpu, HIP_R_32F, N * OH * OW, F_gpu, HIP_R_32F, C * R * S,
//              &beta, BUF2_gpu, HIP_R_32F, N * OH * OW, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);