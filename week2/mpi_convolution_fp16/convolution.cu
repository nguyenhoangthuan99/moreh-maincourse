#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <hip/hip_fp16.h>
#include "convolution.cuh"
#include "util.h"
#include <thread>
#include <hipblas.h>
#include <mpi.h>
#include <mpi-ext.h>
#define BLOCKS 16
#define BLOCK_SIZE 16
static int ngpu;
static half *I_gpu[1024], *F_gpu[1024], *BUF1_gpu[1024];
static float *BUF2_gpu[1024], *O_gpu[1024];
static int N_gpu_start[1024], N_gpu_end[1024];
#define CHECK_CUDA(call)                                                 \
  do                                                                     \
  {                                                                      \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess)                                          \
    {                                                                    \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

__global__ void im2col_kernel(half *_I, half *workspace, int N, int C, int H, int W,
                              int R, int S, int pad_h, int pad_w, int stride_h,
                              int stride_w, int dilation_h, int dilation_w)
{
  const int ON = N;
  const int OH = 1 + (H + 2 * pad_h - (((R - 1) * dilation_h) + 1)) / stride_h;
  const int OW = 1 + (W + 2 * pad_w - (((S - 1) * dilation_w) + 1)) / stride_w;
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  // int ow = tidx % OW;
  // int oh = (tidx) / OW % OH;
  // int on = tidx / OW / OH;

  const int ow = tidx % OW;
  const int oh = (tidx / OW) % OH;
  const int on = tidx / (OH * OW);
  if (ow < OW && oh < OH && on < N)
  {
#pragma unroll
    for (int c = 0; c < C; ++c)
      for (int r = 0; r < R; ++r)
        for (int s = 0; s < S; ++s)
        {
          const int h = oh * stride_h - pad_h + r * dilation_h;
          const int w = ow * stride_w - pad_w + s * dilation_w;
          if (h < 0 || h >= H || w < 0 || w >= W)
            continue;
          // half temp = ;
          // for (int on = 0; on < ON; ++on)
          // {

          workspace[((c * R * S) + (r * S) + s) * (ON * OH * OW) +
                    (on * OH * OW + oh * OW + ow)] =
              _I[on * C * H * W + c * H * W + h * W + w];
          // }
        }
  }
}

__global__ void matmul_gpu(half *_A, half *_B, float *_C, int M, int N, int K, int temp_N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int k;
  __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float A1[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float A2[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float A3[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float A4[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float A5[BLOCK_SIZE][BLOCK_SIZE];

  __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B1[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B2[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B3[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B4[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B5[BLOCK_SIZE][BLOCK_SIZE];

  // float sum[8] = {0.0f};//, sum1 = 0., sum2 = 0., sum3 = 0., sum4 = 0, sum5 = 0, sum6 = 0, sum7 = 0;
  float sum[6][6] = {0};
  float zero_float = 0.;
  // float4 sum2 = make_float4(0, 0, 0, 0);
  int boundk = (K + BLOCK_SIZE - 1) / BLOCK_SIZE;
  int aid = 6 * idy * K + threadIdx.x;
  int bid = threadIdx.y * N + idx * 6;
  // printf("%d %d - %d %d\n", aid, bid, idx, idy);
  for (k = 0; k < boundk; k++)
  {
    As[threadIdx.y][threadIdx.x] = (idy * 6 < M) && k * BLOCK_SIZE + threadIdx.x < K ? __half2float(_A[aid]) : zero_float;
    A1[threadIdx.y][threadIdx.x] = (idy * 6 + 1 < M) && k * BLOCK_SIZE + threadIdx.x < K ? __half2float(_A[aid + K]) : zero_float;
    A2[threadIdx.y][threadIdx.x] = (idy * 6 + 2 < M) && k * BLOCK_SIZE + threadIdx.x < K ? __half2float(_A[aid + 2 * K]) : zero_float;
    A3[threadIdx.y][threadIdx.x] = (idy * 6 + 3 < M) && k * BLOCK_SIZE + threadIdx.x < K ? __half2float(_A[aid + 3 * K]) : zero_float;
    A4[threadIdx.y][threadIdx.x] = (idy * 6 + 4 < M) && k * BLOCK_SIZE + threadIdx.x < K ? __half2float(_A[aid + 4 * K]) : zero_float;
    A5[threadIdx.y][threadIdx.x] = (idy * 6 + 5 < M) && k * BLOCK_SIZE + threadIdx.x < K ? __half2float(_A[aid + 5 * K]) : zero_float;

    Bs[threadIdx.y][threadIdx.x] = (idx * 6 < N) && k * BLOCK_SIZE + threadIdx.y < K ? __half2float(_B[bid]) : zero_float;
    B1[threadIdx.y][threadIdx.x] = (idx * 6 + 1 < N) && k * BLOCK_SIZE + threadIdx.y < K ? __half2float(_B[bid + 1]) : zero_float;
    B2[threadIdx.y][threadIdx.x] = (idx * 6 + 2 < N) && k * BLOCK_SIZE + threadIdx.y < K ? __half2float(_B[bid + 2]) : zero_float;
    B3[threadIdx.y][threadIdx.x] = (idx * 6 + 3 < N) && k * BLOCK_SIZE + threadIdx.y < K ? __half2float(_B[bid + 3]) : zero_float;
    B4[threadIdx.y][threadIdx.x] = (idx * 6 + 4 < N) && k * BLOCK_SIZE + threadIdx.y < K ? __half2float(_B[bid + 4]) : zero_float;
    B5[threadIdx.y][threadIdx.x] = (idx * 6 + 5 < N) && k * BLOCK_SIZE + threadIdx.y < K ? __half2float(_B[bid + 5]) : zero_float;
    __syncthreads();

    // #pragma unroll 32
    for (int ex = 0; ex < BLOCK_SIZE; ex++)
    {
      float a0 = As[threadIdx.y][ex];
      float a1 = A1[threadIdx.y][ex];
      float a2 = A2[threadIdx.y][ex];
      float a3 = A3[threadIdx.y][ex];
      float a4 = A4[threadIdx.y][ex];
      float a5 = A5[threadIdx.y][ex];
      float b0 = Bs[ex][threadIdx.x];
      float b1 = B1[ex][threadIdx.x];
      float b2 = B2[ex][threadIdx.x];
      float b3 = B3[ex][threadIdx.x];
      float b4 = B4[ex][threadIdx.x];
      float b5 = B5[ex][threadIdx.x];

      sum[0][0] += a0 * b0;
      sum[0][1] += a0 * b1;
      sum[0][2] += a0 * b2;
      sum[0][3] += a0 * b3;
      sum[0][4] += a0 * b4;
      sum[0][5] += a0 * b5;

      sum[1][0] += a1 * b0;
      sum[1][1] += a1 * b1;
      sum[1][2] += a1 * b2;
      sum[1][3] += a1 * b3;
      sum[1][4] += a1 * b4;
      sum[1][5] += a1 * b5;

      sum[2][0] += a2 * b0;
      sum[2][1] += a2 * b1;
      sum[2][2] += a2 * b2;
      sum[2][3] += a2 * b3;
      sum[2][4] += a2 * b4;
      sum[2][5] += a2 * b5;

      sum[3][0] += a3 * b0;
      sum[3][1] += a3 * b1;
      sum[3][2] += a3 * b2;
      sum[3][3] += a3 * b3;
      sum[3][4] += a3 * b4;
      sum[3][5] += a3 * b5;

      sum[4][0] += a4 * b0;
      sum[4][1] += a4 * b1;
      sum[4][2] += a4 * b2;
      sum[4][3] += a4 * b3;
      sum[4][4] += a4 * b4;
      sum[4][5] += a4 * b5;

      sum[5][0] += a5 * b0;
      sum[5][1] += a5 * b1;
      sum[5][2] += a5 * b2;
      sum[5][3] += a5 * b3;
      sum[5][4] += a5 * b4;
      sum[5][5] += a5 * b5;
    }
    __syncthreads();
    aid += BLOCK_SIZE;
    bid += BLOCK_SIZE * N;
  }
  // float sum[4] = {sum0, sum1, sum2, sum3};

  int temp_K = M, temp_OHOW = N / temp_N;
  float left = min(6, N - idx * 6);
  float left_j = min(6, M - idy * 6);
#pragma unroll
  for (int j = 0; j < left_j; j++)
  {
#pragma unroll
    for (int i = 0; i < left; i++)
    {
      int c_index = idx * 6 + i + (idy * 6 + j) * N;
      int ohow = c_index % temp_OHOW;
      int n = c_index / temp_OHOW % temp_N;
      k = c_index / temp_OHOW / temp_N;
      int new_c_index = ((n * temp_K + k) * temp_OHOW + ohow);
      _C[new_c_index] = sum[j][i];
    }
  }
}

void convolution_thread(half *_I, half *_F, float *_O, half *_BUF1, float *_BUF2, int N,
                        int C, int H, int W, int K, int R, int S, int pad_h, int pad_w,
                        int stride_h, int stride_w, int dilation_h, int dilation_w, int gpu_id = 0)
{
  // Remove this line after you complete the convolution on GPU
  // naive_cpu_convolution_im2col(_I, _F, _O, _BUF1, _BUF2, N, C, H, W, K, R, S,
  //                              pad_h, pad_w, stride_h, stride_w, dilation_h,
  //                              dilation_w);
  half *I = _I, *F = _F, *BUF1 = _BUF1;
  float *O = _O, *BUF2 = _BUF2;
  hipSetDevice(gpu_id);
  int num_stream_blocks = min(BLOCKS, N);
  hipStream_t data_h2d_stream, data_d2h_stream, calc_im2col_stream, calc_matmul_stream;
  hipStream_t streams[num_stream_blocks];
  hipStreamCreate(&data_h2d_stream);
  // hipStreamCreate(&data_d2h_stream);
  // hipStreamCreate(&calc_im2col_stream);
  // hipStreamCreate(&calc_matmul_stream);
  hipEvent_t events_data[num_stream_blocks], events_im2col_cals[num_stream_blocks], events_matmul_cals[num_stream_blocks];

  for (int i = 0; i < num_stream_blocks; i++)
  {
    hipEventCreate(&events_data[i]);
    hipEventCreate(&events_im2col_cals[i]);
    hipEventCreate(&events_matmul_cals[i]);
    hipStreamCreate(&streams[i]);
  }

  int Nbegin[num_stream_blocks], Nend[num_stream_blocks];
  for (size_t i = 0; i < num_stream_blocks; i++)
  {
    Nbegin[i] = N / num_stream_blocks * i;
    Nend[i] = N / num_stream_blocks * (i + 1);
    if (i == num_stream_blocks - 1)
      Nend[i] = N;
  }

  const int OH = 1 + (H + 2 * pad_h - (((R - 1) * dilation_h) + 1)) / stride_h;
  const int OW = 1 + (W + 2 * pad_w - (((S - 1) * dilation_w) + 1)) / stride_w;

  hipMemcpyAsync(F_gpu[gpu_id], _F, sizeof(half) * K * C * R * S, hipMemcpyHostToDevice, data_h2d_stream); // data_h2d_stream

  for (int i = 0; i < num_stream_blocks; i++)
  {
    hipMemcpyAsync(&I_gpu[gpu_id][Nbegin[i] * C * H * W], &_I[Nbegin[i] * C * H * W], sizeof(half) * (Nend[i] - Nbegin[i]) * C * H * W, hipMemcpyHostToDevice, streams[i]);
    // hipEventRecord(events_data[i], data_h2d_stream);
  }
  hipStreamSynchronize(data_h2d_stream);
  for (int i = 0; i < num_stream_blocks; i++)
  {
    dim3 blockDimIm2Col(640);
    dim3 gridDimIm2Col((OH * OW * (Nend[i] - Nbegin[i]) + blockDimIm2Col.x - 1) / blockDimIm2Col.x);
    // hipStreamWaitEvent(calc_im2col_stream, events_data[i]);
    im2col_kernel<<<gridDimIm2Col, blockDimIm2Col, 0, streams[i]>>>(&I_gpu[gpu_id][Nbegin[i] * C * H * W], &BUF1_gpu[gpu_id][Nbegin[i] * C * R * S * OH * OW],
                                                                    (Nend[i] - Nbegin[i]), C, H, W,
                                                                    R, S, pad_h, pad_w, stride_h,
                                                                    stride_w, dilation_h, dilation_w);
    // hipEventRecord(events_im2col_cals[i], calc_im2col_stream);
    // CHECK_CUDA(hipDeviceSynchronize());
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 gridDim((((Nend[i] - Nbegin[i]) * OH * OW + 5) / 6 + blockDim.x - 1) / blockDim.x, ((K + 5) / 6 + blockDim.y - 1) / blockDim.y, 1);
    // hipStreamWaitEvent(calc_matmul_stream, events_im2col_cals[i]);
    matmul_gpu<<<gridDim, blockDim, 0, streams[i]>>>(F_gpu[gpu_id], &BUF1_gpu[gpu_id][Nbegin[i] * C * R * S * OH * OW],
                                                     &BUF2_gpu[gpu_id][Nbegin[i] * K * OH * OW],
                                                     K, (Nend[i] - Nbegin[i]) * OH * OW, C * R * S,
                                                     (Nend[i] - Nbegin[i]));
    // start reshape GPU
    // dim3 reshape_dimBlock(16, 16);
    // dim3 reshape_dimGrid(((Nend[i] - Nbegin[i]) + reshape_dimBlock.x - 1) / reshape_dimBlock.x, (K + reshape_dimBlock.y - 1) / reshape_dimBlock.y);
    // reshape_gpu<<<reshape_dimGrid, reshape_dimBlock, K * (Nend[i] - Nbegin[i]) * OH * OW * sizeof(float),calc_matmul_stream>>>(&BUF2_gpu[Nbegin[i] * K * OH * OW], &O_gpu[Nbegin[i] * K * OH * OW], (Nend[i] - Nbegin[i]), K, OH, OW);

    //
    // hipEventRecord(events_matmul_cals[i], calc_matmul_stream);

    // hipStreamWaitEvent(data_d2h_stream, events_matmul_cals[i]);
    // hipMemcpyAsync(&_O[Nbegin[i] * K * OH * OW], &BUF2_gpu[gpu_id][Nbegin[i] * K * OH * OW], sizeof(float) * K * (Nend[i] - Nbegin[i]) * OH * OW, hipMemcpyDeviceToHost, streams[i]);//data_d2h_stream
  }
  for (int i = 0; i < num_stream_blocks; i++)
  {
    hipMemcpyAsync(&_O[Nbegin[i] * K * OH * OW], &BUF2_gpu[gpu_id][Nbegin[i] * K * OH * OW], sizeof(float) * K * (Nend[i] - Nbegin[i]) * OH * OW, hipMemcpyDeviceToHost, streams[i]); // data_d2h_stream
  }
  // hipMemcpyAsync(_O, BUF2_gpu[gpu_id], sizeof(float) * K * N * OH * OW, hipMemcpyDeviceToHost);
  CHECK_CUDA(hipDeviceSynchronize());
  // double start = get_time();
  // reshape(BUF2, O, N, K, OH, OW);
  // double end = get_time();
  // printf("\nreshape time: %lf\n", end - start);
  // CHECK_CUDA(hipDeviceSynchronize());
}

void convolution_node(half *_I, half *_F, float *_O, half *_BUF1, float *_BUF2, int N,
                      int C, int H, int W, int K, int R, int S, int pad_h, int pad_w,
                      int stride_h, int stride_w, int dilation_h, int dilation_w)
{
  const int ON = N;
  const int OC = K;
  const int OH = 1 + (H + 2 * pad_h - (((R - 1) * dilation_h) + 1)) / stride_h;
  const int OW = 1 + (W + 2 * pad_w - (((S - 1) * dilation_w) + 1)) / stride_w;
  std::thread threads[ngpu];
  for (int i = 0; i < ngpu; i++)
    threads[i] = std::thread(convolution_thread, &_I[N_gpu_start[i] * C * H * W], _F, &_O[N_gpu_start[i] * K * OH * OW],
                             BUF1_gpu[i],
                             BUF2_gpu[i],
                             (N_gpu_end[i] - N_gpu_start[i]), C, H, W, K, R, S, pad_h, pad_w,
                             stride_h, stride_w, dilation_h, dilation_w, i);
  /* Wait for all threads finish */

  for (int i = 0; i < ngpu; i++)
    threads[i].join();
}
void convolution(half *_I, half *_F, float *_O, half *_BUF1, float *_BUF2, int N,
                 int C, int H, int W, int K, int R, int S, int pad_h, int pad_w,
                 int stride_h, int stride_w, int dilation_h, int dilation_w, int mpi_rank, int mpi_world_size)
{
  const int ON = N;
  const int OC = K;
  const int OH = 1 + (H + 2 * pad_h - (((R - 1) * dilation_h) + 1)) / stride_h;
  const int OW = 1 + (W + 2 * pad_w - (((S - 1) * dilation_w) + 1)) / stride_w;

  MPI_Request request[2];         // = (MPI_Request *)malloc(2 * sizeof(MPI_Request));
  MPI_Status status[2];
  int sendcounts[mpi_world_size]; //= (int *)malloc(mpi_world_size * sizeof(int));
  int recvcounts[mpi_world_size];
  int displ[mpi_world_size]; // = (int *)malloc(mpi_world_size * sizeof(int));
  int displ_O[mpi_world_size];

  int Nbegin[mpi_world_size], Nend[mpi_world_size];
  for (size_t i = 0; i < mpi_world_size; i++)
  {
    Nbegin[i] = N / mpi_world_size * i;
    Nend[i] = N / mpi_world_size * (i + 1);
    if (i == mpi_world_size - 1)
      Nend[i] = N;

    sendcounts[i] = (Nend[i] - Nbegin[i]) * C * H * W * sizeof(half);
    recvcounts[i] = (Nend[i] - Nbegin[i]) * K * OH * OW ;
    displ[i] = Nbegin[i] * C * H * W * sizeof(half);
    displ_O[i] = Nbegin[i] * K * OH * OW ;
  }
  MPI_Ibcast((void *)_F, K * C * R * S * sizeof(half), MPI_BYTE, 0, MPI_COMM_WORLD, request);

  MPI_Iscatterv((void *)_I, sendcounts, displ, MPI_BYTE, (void *)_I, N * C * H * W * sizeof(half), MPI_BYTE, 0, MPI_COMM_WORLD, request + 1);
  N = sendcounts[mpi_rank] / (C * H * W)/ sizeof(half);

  MPI_Request request_result;
  MPI_Waitall(2, request, status);
  
  convolution_node(_I, _F, _O, _BUF1, _BUF2, N,
                   C, H, W, K, R, S, pad_h, pad_w,
                   stride_h, stride_w, dilation_h, dilation_w);
  MPI_Igatherv(_O, N * K * OH * OW , MPI_FLOAT, _O, recvcounts, displ_O, MPI_FLOAT, 0, MPI_COMM_WORLD, &request_result);
  MPI_Wait(&request_result, MPI_STATUS_IGNORE);
}
void convolution_initialize(int N, int C, int H, int W, int K, int R, int S,
                            int pad_h, int pad_w, int stride_h, int stride_w,
                            int dilation_h, int dilation_w)
{
  const int ON = N;
  const int OC = K;
  const int OH = 1 + (H + 2 * pad_h - (((R - 1) * dilation_h) + 1)) / stride_h;
  const int OW = 1 + (W + 2 * pad_w - (((S - 1) * dilation_w) + 1)) / stride_w;
  hipGetDeviceCount(&ngpu);
  // ngpu = 1;
  printf("\nNum GPUs: %d\n", ngpu);
  for (size_t i = 0; i < ngpu; i++)
  {
    N_gpu_start[i] = N / ngpu * i;
    N_gpu_end[i] = N / ngpu * (i + 1);
    if (i == ngpu - 1)
      N_gpu_end[i] = N;
  }

  for (int i = 0; i < ngpu; i++)
  {
    hipSetDevice(i);
    CHECK_CUDA(hipMalloc(&I_gpu[i], sizeof(half) * (N_gpu_end[i] - N_gpu_start[i]) * C * H * W));
    // CHECK_CUDA(hipMalloc(&O_gpu, sizeof(float) * ON * OH * OW * OC));
    CHECK_CUDA(hipMalloc(&F_gpu[i], sizeof(half) * K * C * R * S));
    CHECK_CUDA(hipMalloc(&BUF1_gpu[i], sizeof(half) * C * R * S * (N_gpu_end[i] - N_gpu_start[i]) * OH * OW));
    CHECK_CUDA(hipMalloc(&BUF2_gpu[i], sizeof(float) * K * (N_gpu_end[i] - N_gpu_start[i]) * OH * OW));
  }

  CHECK_CUDA(hipDeviceSynchronize());
}

void convolution_cleanup(half *_I, half *_F, float *_O, int N, int C, int H,
                         int W, int K, int R, int S, int pad_h, int pad_w,
                         int stride_h, int stride_w, int dilation_h,
                         int dilation_w)
{
  hipFree(I_gpu);
  hipFree(F_gpu);
  // hipFree(O_gpu);
  hipFree(BUF1_gpu);
  hipFree(BUF2_gpu);
  CHECK_CUDA(hipDeviceSynchronize());
}
// hipblasHandle_t handle;
// hipblasCreate(&handle);
// const half alpha = 1.0f, beta = 0.0f;
// hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N * OH * OW, K, C * R * S,
//              &alpha, BUF1_gpu, HIP_R_32F, N * OH * OW, F_gpu, HIP_R_32F, C * R * S,
//              &beta, BUF2_gpu, HIP_R_32F, N * OH * OW, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);