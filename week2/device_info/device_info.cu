
#include <hip/hip_runtime.h>
#include <cstdio>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

int main() {
  int count;
  CHECK_CUDA(hipGetDeviceCount(&count));

  printf("Number of devices: %d\n", count);
  hipDeviceProp_t props[4];
  for (int i = 0; i < count; ++i) {
    printf("\tdevice %d:\n", i);
    // TODO: get and print device properties
    hipGetDeviceProperties(&props[i], i);
    printf("\t\t- Device name: %s\n",&props[i].name);
    printf("\t\t- multiProcessorCount: %d\n",props[i].multiProcessorCount);
    printf("\t\t- maxThreadsPerBlock: %d\n",props[i].maxThreadsPerBlock);
    printf("\t\t- totalGlobalMem: %ld Mb\n",props[i].totalGlobalMem/1024/1024);
    printf("\t\t- sharedMemPerBlock: %ld Kb\n",props[i].sharedMemPerBlock/1024);

  }

  return 0;
}
