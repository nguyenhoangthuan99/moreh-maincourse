#include "hip/hip_runtime.h"
#include <cstdio>
#include <thread>
#include "matmul.h"
#include "util.h"
// extern __device__ int g[N];
#define BLOCK_SIZE 32
#define BLOCKS 4
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))
// extern __device__ void vecmul(float *_A, float *_B, int idx, int idy, int K, int N, float *sum);
#define CHECK_CUDA(call)                                                 \
  do                                                                     \
  {                                                                      \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess)                                          \
    {                                                                    \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// Device(GPU) pointers
static int ngpu;
static float *A_gpu[1024], *B_gpu[1024], *C_gpu[1024], *AT_gpu[1024];
static int M_gpu_start[1024], M_gpu_end[1024];

__global__ void cuda_transpose(float *in, float *out)
{
  __shared__ float tile[BLOCK_SIZE][BLOCK_SIZE + 1];
  int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  int y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
  tile[threadIdx.y][threadIdx.x] = in[y * gridDim.x * BLOCK_SIZE + x];
  __syncthreads();
  x = blockIdx.y * BLOCK_SIZE + threadIdx.x;
  y = blockIdx.x * BLOCK_SIZE + threadIdx.y;
  out[y * gridDim.y * BLOCK_SIZE + x] = tile[threadIdx.x][threadIdx.y];
}

__global__ void matmul_tiling(float *A, float *B, float *C, int M, int N, int K)
{
}

__global__ void matmul_gpu(float *_A, float *_B, float *_C, int M, int N, int K)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int k;
  __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B1[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B2[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B3[BLOCK_SIZE][BLOCK_SIZE];
  // __shared__ float B4[BLOCK_SIZE][BLOCK_SIZE];
  // __shared__ float B5[BLOCK_SIZE][BLOCK_SIZE];
  // __shared__ float B6[BLOCK_SIZE][BLOCK_SIZE];
  // __shared__ float B7[BLOCK_SIZE][BLOCK_SIZE];
  float c0;
  // float sum[8] = {0.0f};//, sum1 = 0., sum2 = 0., sum3 = 0., sum4 = 0, sum5 = 0, sum6 = 0, sum7 = 0;
  float4 sum = make_float4(0, 0, 0, 0);
  // float4 sum2 = make_float4(0, 0, 0, 0);
  int boundk = (K + BLOCK_SIZE - 1) / BLOCK_SIZE;
  int aid = idy * K + threadIdx.x;
  int bid = threadIdx.y * N + idx * 4;
  // printf("%d %d - %d %d\n", aid, bid, idx, idy);
  for (k = 0; k < boundk; k++)
  {
    As[threadIdx.y][threadIdx.x] = (idy < M) && k * BLOCK_SIZE + threadIdx.x < K ? _A[aid] : 0;
    Bs[threadIdx.y][threadIdx.x] = (idx * 4 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid] : 0;
    B1[threadIdx.y][threadIdx.x] = (idx * 4 + 1 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid + 1] : 0;
    B2[threadIdx.y][threadIdx.x] = (idx * 4 + 2 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid + 2] : 0;
    B3[threadIdx.y][threadIdx.x] = (idx * 4 + 3 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid + 3] : 0;
    // B4[threadIdx.y][threadIdx.x] = (idx * 8 + 4 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid + 4] : 0;
    // B5[threadIdx.y][threadIdx.x] = (idx * 8 + 5 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid + 5] : 0;
    // B6[threadIdx.y][threadIdx.x] = (idx * 8 + 6 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid + 6] : 0;
    // B7[threadIdx.y][threadIdx.x] = (idx * 8 + 7 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid + 7] : 0;
    __syncthreads();

    // #pragma unroll 32
    for (int e = 0; e < BLOCK_SIZE; e++)
    {
      float a = As[threadIdx.y][e];
      sum = make_float4(sum.x + a * Bs[e][threadIdx.x], sum.y + a * B1[e][threadIdx.x], sum.z + a * B2[e][threadIdx.x], sum.w + a * B3[e][threadIdx.x]);
      // sum2 = make_float4(sum.x + a * B4[e][threadIdx.x], sum.y + a * B5[e][threadIdx.x], sum.z + a * B6[e][threadIdx.x], sum.w + a * B7[e][threadIdx.x]);
      // sum[0] += a * Bs[e][threadIdx.x];
      // sum[1] += a * B1[e][threadIdx.x];
      // sum[2] += a * B2[e][threadIdx.x];
      // sum[3] += a * B3[e][threadIdx.x];
      // sum[4] += a * B4[e][threadIdx.x];
      // sum[5] += a * B5[e][threadIdx.x];
      // sum[6] += a * B6[e][threadIdx.x];
      // sum[7] += a * B7[e][threadIdx.x];
    }
    __syncthreads();
    aid += BLOCK_SIZE;
    bid += BLOCK_SIZE * N;
  }
  if (idy < M)
  {
    // reinterpret_cast<float4 *>(_C)[idx * 8 + idy * N] = sum;
    // reinterpret_cast<float4 *>(_C)[idx * 8 + 4 + idy * N] = sum;
        float left = min(4, N - idx * 4);
    #pragma unroll 4
        for (int i = 0; i < left; i++)
        {
          _C[idx * 4 + i + idy * N] = ((float*)(&sum))[i];
        }
  }
}

void matmul_thread(float *_A, float *_B, float *_C, int M, int N, int K, int gpu_id = 0)
{
  // Remove this line after you complete the matmul on GPU
  // naive_cpu_matmul(_A, _B, _C, M, N, K);

  // (TODO) Upload A and B matrix to GPU
  hipSetDevice(gpu_id);

  hipStream_t data_h2d_stream, data_d2h_stream, calc_stream;
  hipStreamCreate(&data_h2d_stream);
  hipStreamCreate(&data_d2h_stream);
  hipStreamCreate(&calc_stream);
  hipEvent_t events_data[BLOCKS], events_cals[BLOCKS];

  for (int i = 0; i < BLOCKS; i++)
  {
    hipEventCreate(&events_data[i]);
    hipEventCreate(&events_cals[i]);
  }

  int Mbegin[BLOCKS], Mend[BLOCKS];
  for (size_t i = 0; i < BLOCKS; i++)
  {
    Mbegin[i] = M / BLOCKS * i;
    Mend[i] = M / BLOCKS * (i + 1);
    if (i == BLOCKS - 1)
      Mend[i] = M;
  }

  hipMemcpyAsync(B_gpu[gpu_id], _B, sizeof(float) * N * K, hipMemcpyHostToDevice, data_h2d_stream);

  // hipMemcpyAsync(A_gpu, _A, sizeof(float) * M * K, hipMemcpyHostToDevice);
  for (int i = 0; i < BLOCKS; i++)
  {
    hipMemcpyAsync(&A_gpu[gpu_id][Mbegin[i] * K], &_A[Mbegin[i] * K],
                    (Mend[i] - Mbegin[i]) * K * sizeof(float),
                    hipMemcpyHostToDevice, data_h2d_stream);
    hipEventRecord(events_data[i], data_h2d_stream);
  }

  // (TODO) Launch kernel on a GPU
  for (int i = 0; i < BLOCKS; i++)
  {
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 gridDim(((N + 3) / 4 + blockDim.x - 1) / blockDim.x, ((Mend[i] - Mbegin[i]) + blockDim.y - 1) / blockDim.y, 1);
    // dim3 gridDim((N + blockDim.x - 1) / blockDim.x, ((Mend[i] - Mbegin[i]) + blockDim.y - 1) / blockDim.y, 1);

    // printf("\n%d %d\n", ((N + 3) / 4 + blockDim.x - 1) / blockDim.x, ((Mend[i] - Mbegin[i]) + blockDim.y - 1) / blockDim.y);
    hipStreamWaitEvent(calc_stream, events_data[i]);
    matmul_gpu<<<gridDim, blockDim, 0, calc_stream>>>(&A_gpu[gpu_id][Mbegin[i] * K], B_gpu[gpu_id], &C_gpu[gpu_id][Mbegin[i] * N], (Mend[i] - Mbegin[i]), N, K);
    hipEventRecord(events_cals[i], calc_stream);
    hipStreamWaitEvent(data_d2h_stream, events_cals[i]);
    CHECK_CUDA(hipMemcpyAsync(&_C[Mbegin[i] * N], &C_gpu[gpu_id][Mbegin[i] * N], sizeof(float) * N * (Mend[i] - Mbegin[i]), hipMemcpyDeviceToHost, data_d2h_stream));
  }

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
void matmul(float *_A, float *_B, float *_C, int M, int N, int K)
{
  std::thread threads[ngpu];
  for (int i = 0; i < ngpu; i++)
    threads[i] = std::thread(matmul_thread, &_A[M_gpu_start[i] * K], _B, &_C[M_gpu_start[i] * N], M_gpu_end[i] - M_gpu_start[i], N, K, i);
  /* Wait for all threads finish */

  for (int i = 0; i < ngpu; i++)
    threads[i].join();
}
void matmul_init(int M, int N, int K)
{
  // (TODO) Allocate device memory
  hipGetDeviceCount(&ngpu);
  ngpu = 1;
  for (size_t i = 0; i < ngpu; i++)
  {
    M_gpu_start[i] = M / ngpu * i;
    M_gpu_end[i] = M / ngpu * (i + 1);
    if (i == ngpu - 1)
      M_gpu_end[i] = M;
  }
  for (int i = 0; i < ngpu; i++)
  {
    hipSetDevice(i);
    CHECK_CUDA(hipMalloc(&A_gpu[i], (M_gpu_end[i] - M_gpu_start[i]) * K * sizeof(float)));
    // CHECK_CUDA(hipMalloc(&AT_gpu[i], (M_gpu_end[i] - M_gpu_start[i]) * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&B_gpu[i], K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&C_gpu[i], (M_gpu_end[i] - M_gpu_start[i]) * N * sizeof(float)));
  }

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_cleanup(float *_A, float *_B, float *_C, int M, int N, int K)
{
  // (TODO) Do any post-matmul cleanup work here.
  hipFree(A_gpu);
  // hipFree(AT_gpu);
  hipFree(B_gpu);
  hipFree(C_gpu);
  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
