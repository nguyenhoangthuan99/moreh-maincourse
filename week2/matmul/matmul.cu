#include "hip/hip_runtime.h"
#include <cstdio>
#include <thread>
#include "matmul.h"
#include "util.h"
// extern __device__ int g[N];
#define BLOCK_SIZE 16
#define BLOCKS 4
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))
// extern __device__ void vecmul(float *_A, float *_B, int idx, int idy, int K, int N, float *sum);
#define CHECK_CUDA(call)                                                 \
  do                                                                     \
  {                                                                      \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess)                                          \
    {                                                                    \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// Device(GPU) pointers
static int ngpu;
static float *A_gpu[1024], *B_gpu[1024], *C_gpu[1024], *AT_gpu[1024];
static int M_gpu_start[1024], M_gpu_end[1024];

__global__ void cuda_transpose(float *in, float *out)
{
  __shared__ float tile[BLOCK_SIZE][BLOCK_SIZE + 1];
  int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  int y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
  tile[threadIdx.y][threadIdx.x] = in[y * gridDim.x * BLOCK_SIZE + x];
  __syncthreads();
  x = blockIdx.y * BLOCK_SIZE + threadIdx.x;
  y = blockIdx.x * BLOCK_SIZE + threadIdx.y;
  out[y * gridDim.y * BLOCK_SIZE + x] = tile[threadIdx.x][threadIdx.y];
}

__global__ void matmul_tiling(float *A, float *B, float *C, int M, int N, int K)
{
}

__global__ void matmul_gpu(float *_A, float *_B, float *_C, int M, int N, int K)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int k;
  __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float A1[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float A2[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float A3[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float A4[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float A5[BLOCK_SIZE][BLOCK_SIZE];

  __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B1[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B2[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B3[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B4[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B5[BLOCK_SIZE][BLOCK_SIZE];

  // float sum[8] = {0.0f};//, sum1 = 0., sum2 = 0., sum3 = 0., sum4 = 0, sum5 = 0, sum6 = 0, sum7 = 0;
  float sum[6][6] = {0};
  float zero_float = 0.;
  // float4 sum2 = make_float4(0, 0, 0, 0);
  int boundk = (K + BLOCK_SIZE - 1) / BLOCK_SIZE;
  int aid = 6 * idy * K + threadIdx.x;
  int bid = threadIdx.y * N + idx * 6;
  // printf("%d %d - %d %d\n", aid, bid, idx, idy);
  for (k = 0; k < boundk; k++)
  {
    As[threadIdx.y][threadIdx.x] = (idy * 6 < M) && k * BLOCK_SIZE + threadIdx.x < K ? _A[aid] : zero_float;
    A1[threadIdx.y][threadIdx.x] = (idy * 6 + 1 < M) && k * BLOCK_SIZE + threadIdx.x < K ? _A[aid + K] : zero_float;
    A2[threadIdx.y][threadIdx.x] = (idy * 6 + 2 < M) && k * BLOCK_SIZE + threadIdx.x < K ? _A[aid + 2 * K] : zero_float;
    A3[threadIdx.y][threadIdx.x] = (idy * 6 + 3 < M) && k * BLOCK_SIZE + threadIdx.x < K ? _A[aid + 3 * K] : zero_float;
    A4[threadIdx.y][threadIdx.x] = (idy * 6 + 4 < M) && k * BLOCK_SIZE + threadIdx.x < K ? _A[aid + 4 * K] : zero_float;
    A5[threadIdx.y][threadIdx.x] = (idy * 6 + 5 < M) && k * BLOCK_SIZE + threadIdx.x < K ? _A[aid + 5 * K] : zero_float;

    Bs[threadIdx.y][threadIdx.x] = (idx * 6 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid] : zero_float;
    B1[threadIdx.y][threadIdx.x] = (idx * 6 + 1 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid + 1] : zero_float;
    B2[threadIdx.y][threadIdx.x] = (idx * 6 + 2 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid + 2] : zero_float;
    B3[threadIdx.y][threadIdx.x] = (idx * 6 + 3 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid + 3] : zero_float;
    B4[threadIdx.y][threadIdx.x] = (idx * 6 + 4 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid + 4] : zero_float;
    B5[threadIdx.y][threadIdx.x] = (idx * 6 + 5 < N) && k * BLOCK_SIZE + threadIdx.y < K ? _B[bid + 5] : zero_float;
    __syncthreads();

    // #pragma unroll 32
    for (int ex = 0; ex < BLOCK_SIZE; ex++)
    {
      float a0 = As[threadIdx.y][ex];
      float a1 = A1[threadIdx.y][ex];
      float a2 = A2[threadIdx.y][ex];
      float a3 = A3[threadIdx.y][ex];
      float a4 = A4[threadIdx.y][ex];
      float a5 = A5[threadIdx.y][ex];
      float b0 = Bs[ex][threadIdx.x];
      float b1 = B1[ex][threadIdx.x];
      float b2 = B2[ex][threadIdx.x];
      float b3 = B3[ex][threadIdx.x];
      float b4 = B4[ex][threadIdx.x];
      float b5 = B5[ex][threadIdx.x];

      sum[0][0] += a0 * b0;
      sum[0][1] += a0 * b1;
      sum[0][2] += a0 * b2;
      sum[0][3] += a0 * b3;
      sum[0][4] += a0 * b4;
      sum[0][5] += a0 * b5;

      sum[1][0] += a1 * b0;
      sum[1][1] += a1 * b1;
      sum[1][2] += a1 * b2;
      sum[1][3] += a1 * b3;
      sum[1][4] += a1 * b4;
      sum[1][5] += a1 * b5;

      sum[2][0] += a2 * b0;
      sum[2][1] += a2 * b1;
      sum[2][2] += a2 * b2;
      sum[2][3] += a2 * b3;
      sum[2][4] += a2 * b4;
      sum[2][5] += a2 * b5;

      sum[3][0] += a3 * b0;
      sum[3][1] += a3 * b1;
      sum[3][2] += a3 * b2;
      sum[3][3] += a3 * b3;
      sum[3][4] += a3 * b4;
      sum[3][5] += a3 * b5;

      sum[4][0] += a4 * b0;
      sum[4][1] += a4 * b1;
      sum[4][2] += a4 * b2;
      sum[4][3] += a4 * b3;
      sum[4][4] += a4 * b4;
      sum[4][5] += a4 * b5;

      sum[5][0] += a5 * b0;
      sum[5][1] += a5 * b1;
      sum[5][2] += a5 * b2;
      sum[5][3] += a5 * b3;
      sum[5][4] += a5 * b4;
      sum[5][5] += a5 * b5;
    }
    __syncthreads();
    aid += BLOCK_SIZE;
    bid += BLOCK_SIZE * N;
  }
  // float sum[4][4] = {{sum00, sum01, sum02, sum03}, {sum10, sum11, sum12, sum13}, {sum20, sum21, sum22, sum23}, {sum30, sum31, sum32, sum33}};
  // reinterpret_cast<float4 *>(_C)[idx * 8 + idy * N] = sum;
  // reinterpret_cast<float4 *>(_C)[idx * 8 + 4 + idy * N] = sum;
  float left = min(6, N - idx * 6);
  float left_j = min(6, M - idy * 6);
  #pragma unroll 
  for (int j = 0; j < left_j; j++)
  {
    #pragma unroll
    for (int i = 0; i < left; i++)
      _C[idx * 6 + i + (idy * 6 + j) * N] = sum[j][i];
  }
}

void matmul_thread(float *_A, float *_B, float *_C, int M, int N, int K, int gpu_id = 0)
{
  // Remove this line after you complete the matmul on GPU
  // naive_cpu_matmul(_A, _B, _C, M, N, K);

  // (TODO) Upload A and B matrix to GPU
  double total = 0.;
  hipSetDevice(gpu_id);

  hipStream_t data_h2d_stream, data_d2h_stream, calc_stream;
  hipStreamCreate(&data_h2d_stream);
  hipStreamCreate(&data_d2h_stream);
  hipStreamCreate(&calc_stream);
  hipEvent_t events_data[BLOCKS], events_cals[BLOCKS];

  for (int i = 0; i < BLOCKS; i++)
  {
    hipEventCreate(&events_data[i]);
    hipEventCreate(&events_cals[i]);
  }

  int Mbegin[BLOCKS], Mend[BLOCKS];
  for (size_t i = 0; i < BLOCKS; i++)
  {
    Mbegin[i] = M / BLOCKS * i;
    Mend[i] = M / BLOCKS * (i + 1);
    if (i == BLOCKS - 1)
      Mend[i] = M;
  }

  hipMemcpyAsync(B_gpu[gpu_id], _B, sizeof(float) * N * K, hipMemcpyHostToDevice, data_h2d_stream);

  // hipMemcpyAsync(A_gpu, _A, sizeof(float) * M * K, hipMemcpyHostToDevice);
  for (int i = 0; i < BLOCKS; i++)
  {
    hipMemcpyAsync(&A_gpu[gpu_id][Mbegin[i] * K], &_A[Mbegin[i] * K],
                    (Mend[i] - Mbegin[i]) * K * sizeof(float),
                    hipMemcpyHostToDevice, data_h2d_stream);
    hipEventRecord(events_data[i], data_h2d_stream);
  }

  // (TODO) Launch kernel on a GPU
  for (int i = 0; i < BLOCKS; i++)
  {
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 gridDim(((N + 5) / 6 + blockDim.x - 1) / blockDim.x, (((Mend[i] - Mbegin[i]) + 5) / 6 + blockDim.y - 1) / blockDim.y, 1);
    // dim3 gridDim((N + blockDim.x - 1) / blockDim.x, ((Mend[i] - Mbegin[i]) + blockDim.y - 1) / blockDim.y, 1);

    // printf("\n%d %d\n", ((N + 3) / 4 + blockDim.x - 1) / blockDim.x, ((Mend[i] - Mbegin[i]) + blockDim.y - 1) / blockDim.y);
    hipStreamWaitEvent(calc_stream, events_data[i]);
    // double start = get_time();
    matmul_gpu<<<gridDim, blockDim, 0, calc_stream>>>(&A_gpu[gpu_id][Mbegin[i] * K], B_gpu[gpu_id], &C_gpu[gpu_id][Mbegin[i] * N], (Mend[i] - Mbegin[i]), N, K);
    // CHECK_CUDA(hipDeviceSynchronize());
    // double end = get_time();
    // total += end-start;
    hipEventRecord(events_cals[i], calc_stream);
    hipStreamWaitEvent(data_d2h_stream, events_cals[i]);
    CHECK_CUDA(hipMemcpyAsync(&_C[Mbegin[i] * N], &C_gpu[gpu_id][Mbegin[i] * N], sizeof(float) * N * (Mend[i] - Mbegin[i]), hipMemcpyDeviceToHost, data_d2h_stream));
  }

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
  // printf("\n Gflop only calculate: %lf \n",2.0 * M * N * K / total / 1e9);
}
void matmul(float *_A, float *_B, float *_C, int M, int N, int K)
{
  std::thread threads[ngpu];
  for (int i = 0; i < ngpu; i++)
    threads[i] = std::thread(matmul_thread, &_A[M_gpu_start[i] * K], _B, &_C[M_gpu_start[i] * N], M_gpu_end[i] - M_gpu_start[i], N, K, i);
  /* Wait for all threads finish */

  for (int i = 0; i < ngpu; i++)
    threads[i].join();
}
void matmul_init(int M, int N, int K)
{
  // (TODO) Allocate device memory
  hipGetDeviceCount(&ngpu);
  ngpu = 1;
  for (size_t i = 0; i < ngpu; i++)
  {
    M_gpu_start[i] = M / ngpu * i;
    M_gpu_end[i] = M / ngpu * (i + 1);
    if (i == ngpu - 1)
      M_gpu_end[i] = M;
  }
  for (int i = 0; i < ngpu; i++)
  {
    hipSetDevice(i);
    CHECK_CUDA(hipMalloc(&A_gpu[i], (M_gpu_end[i] - M_gpu_start[i]) * K * sizeof(float)));
    // CHECK_CUDA(hipMalloc(&AT_gpu[i], (M_gpu_end[i] - M_gpu_start[i]) * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&B_gpu[i], K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&C_gpu[i], (M_gpu_end[i] - M_gpu_start[i]) * N * sizeof(float)));
  }

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_cleanup(float *_A, float *_B, float *_C, int M, int N, int K)
{
  // (TODO) Do any post-matmul cleanup work here.
  hipFree(A_gpu);
  // hipFree(AT_gpu);
  hipFree(B_gpu);
  hipFree(C_gpu);
  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
