#include "hip/hip_runtime.h"
#include <cstdio>

#include "image_rotation.h"

#define CHECK_CUDA(call)                                                 \
  do                                                                     \
  {                                                                      \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess)                                          \
    {                                                                    \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// Device(GPU) pointers
static float *input_images_gpu, *output_images_gpu;

void rotate_image_naive(float *input_images, float *output_images, int W, int H,
                        float sin_theta, float cos_theta, int num_src_images)
{
  float x0 = W / 2.0f;
  float y0 = H / 2.0f;

  // Rotate images
  for (int i = 0; i < num_src_images; i++)
  {
    for (int dest_x = 0; dest_x < W; dest_x++)
    {
      for (int dest_y = 0; dest_y < H; dest_y++)
      {
        float xOff = dest_x - x0;
        float yOff = dest_y - y0;
        int src_x = (int)(xOff * cos_theta + yOff * sin_theta + x0);
        int src_y = (int)(yOff * cos_theta - xOff * sin_theta + y0);
        if ((src_x >= 0) && (src_x < W) && (src_y >= 0) && (src_y < H))
        {
          output_images[i * H * W + dest_y * W + dest_x] =
              input_images[i * H * W + src_y * W + src_x];
        }
        else
        {
          output_images[i * H * W + dest_y * W + dest_x] = 0.0f;
        }
      }
    }
  }
}

__global__ void rotate_image_device(float *input_images, float *output_images, int W, int H,
                                    float sin_theta, float cos_theta, int num_src_images)
{
  // int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  // int idx = tidx % W;
  // int idy = tidx / W % H;
  // int idz = tidx / W / H;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int idz = blockIdx.z * blockDim.z + threadIdx.z;
  float x0 = W / 2.0f;
  float y0 = H / 2.0f;

  // Rotate images
  float new_px =0.0f;
  float xOff = idx - x0;
  float yOff = idy - y0;
  int src_x = (int)(xOff * cos_theta + yOff * sin_theta + x0);
  int src_y = (int)(yOff * cos_theta - xOff * sin_theta + y0);
  if ((src_x >= 0) && (src_x < W) && (src_y >= 0) && (src_y < H))
  {
    new_px = input_images[idz * H * W + src_y * W + src_x];
  }
  else
  {
    new_px = 0.0f;
  }
  output_images[idz * H * W + idy * W + idx] = new_px;
}

void rotate_image(float *input_images, float *output_images, int W, int H,
                  float sin_theta, float cos_theta, int num_src_images)
{
  // Remove this line after you complete the image rotation on GPU
  // rotate_image_naive(input_images, output_images, W, H, sin_theta, cos_theta,
  //                    num_src_images);

  // (TODO) Upload input images to GPU
  CHECK_CUDA(hipMemcpy(input_images_gpu, input_images, sizeof(float) * W * H * num_src_images, hipMemcpyHostToDevice));

  dim3 blockDim(16,16,1);
  dim3 gridDim(std::ceil(((float)W) / blockDim.x), std::ceil(((float)H) / blockDim.x),num_src_images);
  // dim3 gridDim((num_src_images * H * W + 1023) / 1024);
  // (TODO) Launch kernel on GPU
  rotate_image_device<<<gridDim, blockDim>>>(input_images_gpu, output_images_gpu, W, H, sin_theta, cos_theta, num_src_images);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());
  // (TODO) Download output images from GPU
  CHECK_CUDA(hipMemcpyAsync(output_images, output_images_gpu, sizeof(float) * W * H * num_src_images, hipMemcpyDeviceToHost));
  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_init(int image_width, int image_height, int num_src_images)
{
  // (TODO) Allocate device memory

  CHECK_CUDA(hipMalloc(&input_images_gpu, sizeof(float) * image_height * image_width * num_src_images));
  CHECK_CUDA(hipMalloc(&output_images_gpu, sizeof(float) * image_height * image_width * num_src_images));

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_cleanup()
{
  // (TODO) Free device memory
  hipFree(input_images_gpu);
  hipFree(output_images_gpu);
  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}
